#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>

__global__ void matVecMul(const float* A, const float* B, float* C, const int N, const int M) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < N) {
        float sum = 0.0f;
        
        for(int j = 0; j < M; ++j) {
            sum += A[i * N + j] * B[j];
        }
        C[i] = sum;
        // printf("%.2f\n", sum);
    }
}



int main() {

    hipEvent_t start, stop;
    float milliseconds = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int N = 1024, M = 768;
        
    float* A, *B, *C;

    A = new float[N * M];
    B = new float[M];
    C = new float[N];

    for(int i = 0; i < N; ++i) {
        C[i] = 0.0f;
        for(int j = 0; j < M; ++j) {
            A[i * M + j] = rand() % 100;
        }
    }

    for(int i = 0; i < M; ++i) {
        B[i] = rand() % 100;
    }

    float* d_A, *d_B, *d_C;

    hipMalloc(&d_A, N * M * sizeof(float));
    hipMalloc(&d_B, M * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    hipMemcpy(d_A, A, N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, M *sizeof(float), hipMemcpyHostToDevice);

    const int blockSize = 256;
    const int gridSize = (N + blockSize - 1) / blockSize;    
    
    hipEventRecord(start); // Start recording

    // Contain in anonymous scope
    {

        matVecMul<<<gridSize, blockSize>>>(d_A, d_B, d_C, N, M);

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
            // Handle the error (e.g., exit)
            // Added this because kernel was silently not running and failing because of invald configuration argument meaning that I was giving block threads > max block threads for my GPU
        }
        
        hipDeviceSynchronize();
    }

    hipEventRecord(stop); // Stop recording
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Kernel execution time: %f ms\n", milliseconds);

    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] A;
    A = nullptr;
    delete[] B;
    B = nullptr;
    delete[] C;
    C = nullptr;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}